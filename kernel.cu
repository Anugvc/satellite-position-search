#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "common_include.h"

extern "C" void createPositionVector(sat_pos_st  *d_sat_pos, satellite_st *d_satllites, int numSatellites, int time_mins);
extern "C" void transformlla(sat_pos_st * d_sat_pos, int numSatellites, int time_mins);



__device__ int global_head = -1;
__device__ int global_tail = -1;
__device__ int global_free = 0;

// Helper function to initialize the linked list nodes
__global__ void initLinkedListNodes(LinkedListNode* nodes, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        nodes[idx].next = idx + 1;
    }
    if (idx == n - 1) {
        nodes[idx].next = -1;
    }
}

__device__ int allocateNode(LinkedListNode* nodes, int& free_list) {
    int new_node = -1;
    if (free_list != -1) {
        new_node = free_list;
        free_list = nodes[free_list].next;
    }
    return new_node;
}

__device__ void addNode(LinkedListNode* nodes, sat_pos_st data, int& head, int& tail, int& free_list) {
    int new_node = allocateNode(nodes, free_list);
    if (new_node != -1) {
        nodes[new_node].data = data;
        nodes[new_node].next = -1;

        if (tail != -1) {
            nodes[tail].next = new_node;
        }
        else {
            head = new_node;
        }
        tail = new_node;
    }
}

// CUDA kernel to search for satellites and store the indexes of satellites found, inside the structure array 'searchedSatData'
__global__ void searchSatellitesKernel(sat_pos_st* satelliteData, int numSatellites, int numMins, sat_pos_st* searchedSatData, point_st* region, unsigned  int* d_numSatInsideRegion) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numSatellites * numMins) {
        //int satelliteID = satelliteData[idx].idx;
        //for (int i = 0; i < numSearchIDs; ++i) {
            
            int numPoints = 4; // Rectangle
            bool inside = false;

            double x = satelliteData[idx ].llh[1]; // longitude 
            double y = satelliteData[idx ].llh[0]; // latitude

            double p1x = region[0].longitude;
            double p1y = region[0].latitude;

            /*for (int i = 1; i < numPoints; ++i) {
                double p2x = region[i].longitude;
                double p2y = region[i].latitude;
                if (y > fmin(p1y, p2y)) {
                    if (y <= fmax(p1y, p2y)) {
                        if (x <= fmax(p1x, p2x)) {
                            double xinters;
                            if (p1y != p2y) {
                                xinters = (y - p1y) * (p2x - p1x) / (p2y - p1y) + p1x;
                            }
                            if (p1x == p2x || x <= xinters) {
                                inside = !inside;
                            }
                        }
                    }
                }
                p1x = p2x;
                p1y = p2y;
            }*/
            

            for (int i = 0, j = numPoints - 1; i < numPoints; j = i++) {
                double xi = region[i].longitude, yi = region[i].latitude;
                double xj = region[j].longitude, yj = region[j].latitude;

                bool intersect = ((yi > y) != (yj > y)) && (x < (xj - xi) * (y - yi) / (yj - yi) + xi);
                if (intersect) {
                    inside = !inside;
                }
            }

            /*if (idx == 0)
            {
                printf("lat[0]: %f long[0]: %f \n", satelliteData[idx].llh[0], satelliteData[idx].llh[1]);
            }*/

            //satelliteData[idx * numSearchIDs + i].isInsideRegion = inside;

            if (inside) // The position is inside the given region
            {
                int searchidx = atomicInc(d_numSatInsideRegion, 0xffffffff); // Will set to 0 if value > 0xffffff otherwise increment by 1
                searchedSatData[searchidx].idx = idx; //satelliteData[idx]; // store the result in searchedData array;
                //d_indexArray[*d_numSatInsideRegion]
                //if(idx > 10000 && idx < 10010)
                 //   printf("dnumsat = %u\n", *d_numSatInsideRegion);

                
            }
           
        //}
    }
    //__syncthreads();

    
}
/*
class Point {
public:
    double latitude;
    double longitude;

    Point(double lat, double lon) : latitude(lat), longitude(lon) {}
};

bool isPointInRegion(const std::vector<Point>& polygon, const Point& point) {
    
}
*/


extern "C" unsigned int searchSatellites(sat_pos_st * satelliteData, int numSatellite, int numMins, sat_pos_st* searchedSatData, point_st* region, unsigned int * numSatInsideRegion) {
    

    // Allocate host memory
    //SatelliteData* h_satelliteData = new SatelliteData[numSatellites];
    //int* h_searchIDs = new int[numSearchIDs];
    //LinkedListNode* h_nodes = new LinkedListNode[numSatellites];

    // Initialize host memory
    /*for (int i = 0; i < numSatellites; ++i) {
        h_satelliteData[i] = satelliteArray[i];
    }
    for (int i = 0; i < numSearchIDs; ++i) {
        h_searchIDs[i] = searchIDs[i];
    }
    */

    sat_pos_st* d_searchedSatData, * d_satelliteData;
    hipError_t cudaStatus;

    std::cout << "Inside Kernel.." << std::endl;
    // Allocate device memory for searched satellite data
    cudaStatus = hipMalloc((void**)&d_searchedSatData, sizeof(sat_pos_st) * numSatellite * numMins + 1);
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed in d_searchedSatData!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    //ALlocate device memory for satellite positions and copy satellite date from host to device
    cudaStatus = hipMalloc((void**)&d_satelliteData, sizeof(sat_pos_st) * numSatellite * numMins);
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed in d_satelliteData!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }

    /*for (int i = 0; i < 5000; i = i + 1440)
    {
        std::cout << "inside kernel: " << satelliteData[i].llh[0] << std::endl;
    }*/
    cudaStatus = hipMemcpy(d_satelliteData, satelliteData, sizeof(sat_pos_st) * numSatellite * numMins, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed in satellitedata to d_satellitedata!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    /*cudaStatus = hipMemcpy(satelliteData, d_satelliteData, sizeof(sat_pos_st) * numSatellite * numMins, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed in satellitedata to d_satellitedata!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    for (int i = 0; i < 5000; i = i + 1440)
    {
        std::cout << "inside kernel AFTER GPU COPY: " << satelliteData[i].llh[0] << std::endl;
    }*/

    // Allocate device memory for region to search and copy region from host to device
    point_st* d_region;
    cudaStatus = hipMalloc((void**)&d_region, 4 * sizeof(point_st));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed in d_region!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    cudaStatus = hipMemcpy(d_region, region, 4 * sizeof(point_st), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed in region to d_region!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    unsigned int numsat = 0; // Initial value
    unsigned int* d_numSatInsideRegion; // Variable to store the total number of satellite positions found inside the region (will get incremented atomically inside the kernel)
    cudaStatus = hipMalloc((void **)&d_numSatInsideRegion, sizeof(unsigned  int));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed in d_numSatInsideRegion!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    cudaStatus = hipMemcpy(d_numSatInsideRegion, &numsat, sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed in d_numSatInsideRegion!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    hipDeviceSynchronize();
    //std::cout << "Numsat = " << numsat << std::endl;

    // Copy data from host to device
    //hipMemcpy(d_satelliteData, h_satelliteData, numSatellites * sizeof(SatelliteData), hipMemcpyHostToDevice);
    //hipMemcpy(d_searchIDs, h_searchIDs, numSearchIDs * sizeof(int), hipMemcpyHostToDevice);

    // Initialize the linked list nodes
    int blockSize = 1023;
    int numBlocks = ((numSatellite * numMins) + blockSize - 1) / blockSize;
    //initLinkedListNodes << <numBlocks, blockSize >> > (d_nodes, numSatellites);
    //hipDeviceSynchronize();

    //unsigned int* h_numSat = (unsigned int*)malloc(sizeof(unsigned int));
    //hipMemcpy(h_numSat, d_numSatInsideRegion, sizeof(unsigned int), hipMemcpyDeviceToHost);

    //std::cout << "Total number of Satellite positions before calling kernel " << *h_numSat << std::endl;
    int* d_indexArray;
    hipMalloc((void**)&d_indexArray, sizeof(int) * numSatellite * numMins);


    // Launch the kernel
    searchSatellitesKernel << <numBlocks, blockSize >> > (d_satelliteData, numSatellite, numMins, d_searchedSatData, d_region, d_numSatInsideRegion);
    hipDeviceSynchronize();

    unsigned int*  h_numSat = (unsigned int*) malloc(sizeof(unsigned int));
    *h_numSat = 6;

    cudaStatus = hipMemcpy(numSatInsideRegion, d_numSatInsideRegion, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed for d_numSatInsideRegion to numSatInsideRegion!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;
        
    }
    hipFree(d_satelliteData);
    hipFree(d_region);

    std::cout << "Total number of Satellite positions inside the region: " << *numSatInsideRegion << std::endl;
    //*numSatInsideRegion = *h_numSat;

    std::cout << "Allocating memory: " << std::endl;
    searchedSatData = (sat_pos_st*)malloc(sizeof(sat_pos_st) * (*numSatInsideRegion) + 1);
    cudaStatus = hipMemcpy(searchedSatData, d_searchedSatData, sizeof(sat_pos_st) * 10, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed for d_satelliteData to d_satelliteData!\n");
        std::cout << hipGetErrorString(cudaStatus) << std::endl;

    }
    if(*numSatInsideRegion != 0)
    /*for (int i = 1; i < 20; i++)
    {
        std::cout << "GPU RESULT: " << searchedSatData[i].idx << " " << searchedSatData[i].idx << " " << searchedSatData[i].idx << std::endl;
    }*/

    return (*numSatInsideRegion);

    // Copy result from device to host
    /*hipMemcpy(h_nodes, d_nodes, numSatellites * sizeof(LinkedListNode), hipMemcpyDeviceToHost);

    // Reconstruct the linked list on the host
    int head;
    hipMemcpyFromSymbol(&head, HIP_SYMBOL(global_head), sizeof(int), 0, hipMemcpyDeviceToHost);
    while (head != -1) {
        linkedList.push_back(h_nodes[head]);
        head = h_nodes[head].next;
    }

    // Free device memory
    hipFree(d_satelliteData);
    hipFree(d_searchIDs);
    hipFree(d_nodes);

    // Free host memory
    delete[] h_satelliteData;
    delete[] h_searchIDs;
    delete[] h_nodes;*/


}

/*
// CUDA kernel to populate the result array with satellite info for each minute of the day
__global__ void populateResults(const satellite_st* satellites, sat_pos_st* sat_pos, int numSatellites,int time_mins) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalMinutes = 1440; // Number of minutes in a day

    if (idx < numSatellites * totalMinutes) {
        int satIndex = idx / totalMinutes;
        int minuteOfDay = idx % totalMinutes;

        sat_pos[idx].idx = idx;
        sat_pos[idx].minuteOfDay = minuteOfDay;

        // Perform any additional calculations and populate other fields in ResultInfo as necessary
    }
}

// CUDA kernel to transform satellite ecef to lla
__global__ void transformllaKernel( sat_pos_st* sat_pos, int numSatellites, int time_mins) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalMinutes = 1440; // Number of minutes in a day

    if (idx < numSatellites * totalMinutes) {
        int satIndex = idx / totalMinutes;
        int minuteOfDay = idx % totalMinutes;

        sat_pos[idx].idx = idx;
        sat_pos[idx].minuteOfDay = minuteOfDay;

        // Perform any additional calculations and populate other fields in ResultInfo as necessary
    }
}

extern "C" void createPositionVector(sat_pos_st *d_sat_pos, satellite_st *d_satllites, int numSatellites, int time_mins)
{
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numSatellites * time_mins + threadsPerBlock - 1) / threadsPerBlock;

    populateResults<<<blocksPerGrid, threadsPerBlock >>>(d_satllites, d_sat_pos, numSatellites, time_mins);
    hipDeviceSynchronize();
}

extern "C" void transformlla(sat_pos_st * d_sat_pos,  int numSatellites, int time_mins)
{
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numSatellites * time_mins + threadsPerBlock - 1) / threadsPerBlock;

    transformllaKernel << <blocksPerGrid, threadsPerBlock >> > (d_sat_pos, numSatellites, time_mins);

    hipDeviceSynchronize();
}





hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
*/

/*int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}*/

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
